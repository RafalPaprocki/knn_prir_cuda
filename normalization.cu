#include "hip/hip_runtime.h"
#include "CsvFunctions.h"

void normalize(string outFile, vector< pair<string, vector<double>> >& data);
double findMin(vector<double> data);
double findMax(vector<double> data);

__global__ void normalizeCuda(double *data, int *size, double *min, double *max) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    while (idx < *size) {
        if (idx < *size){
            data[idx] = (data[idx] - *min) / (*max - *min);
        }
        idx += blockDim.x * gridDim.x;
    }
}

int main(int argc, char* argv[])
{	
    vector<pair<string, vector<double>> > out;
    out = readFromCsv("dataset/bigheartdata.csv");
    normalize("normalized_cuda.csv", out);
}


void normalize(string outFile, vector< pair<string, vector<double>> >& data) {
    double min;
    double max;	
    int size = data[0].second.size();
    double * tempData = new double[size];
    for (int i = 0; i < data.size(); ++i) {
        if (data[i].first != "target") {
            min = findMin(data[i].second);
            max = findMax(data[i].second);
            
            //copy vector to array
            for (int j = 0; j < size; ++j){
                tempData[j] = data[i].second[j];
            }

            //creating cuda variables and memory allocation
            double * cudaData;
		    double * cudaMin;
            double * cudaMax;
		    int * dataSize;
            hipMalloc( (void**)&cudaData, sizeof(double) * size ) ;    
    	    hipMalloc( (void**)&cudaMin, sizeof(double));
    	    hipMalloc( (void**)&cudaMax, sizeof(double));
		    hipMalloc( (void**)&dataSize, sizeof(int));

            hipMemcpy( dataSize, &size, sizeof(int), hipMemcpyHostToDevice);
  	        hipMemcpy( cudaData, tempData, sizeof(double) * size, hipMemcpyHostToDevice );
    	    hipMemcpy( cudaMin, &min, sizeof(double), hipMemcpyHostToDevice );
    	    hipMemcpy( cudaMax, &max, sizeof(double), hipMemcpyHostToDevice );
            //normalizowanie
            int num_blocks = ceil(size / 1000) + 1;
            normalizeCuda<<<num_blocks, 1000>>>(cudaData, dataSize, cudaMin, cudaMax);
            
            //copy arra preprocessed by cuda
            hipMemcpy( tempData, cudaData, sizeof(double) * size, hipMemcpyDeviceToHost );

            //copy table to vector 
            for (int j = 0; j < size; ++j){
                data[i].second[j] = tempData[j];
            }

            hipFree( cudaData );
            hipFree( cudaMin );
            hipFree( cudaMax );
            hipFree( dataSize );
            hipDeviceSynchronize();
        }
    }
    delete tempData;
	
    // zapisywanie do pliku
    //writeToCsv(outFile, data);
}

double findMin(vector<double> data) {
    double min = 10000000;
    for (int i = 0; i < data.size(); ++i) {
	
        if (data[i] < min) {
            min = data[i];
        }
    }
    return min;
}

double findMax(vector<double> data) {
    double max = -10000000;
    for (int i = 0; i < data.size(); ++i) {
	
        if (data[i] > max) {
            max = data[i];
        }
    }
    return max;
}

