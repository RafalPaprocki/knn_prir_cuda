#include "hip/hip_runtime.h"
#include "CsvFunctions.h"

double findAverage(vector<double> data);
double findDeviation(vector<double> data);
void standardize(string outFile, vector< pair<string, vector<double>> >& data);

__global__ void standarizeCuda(double *data, double *deviation, double *avg, int *size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    while (idx < *size) {
        if (idx < *size){
            int tempIdx = idx % 13;
            data[idx] = (data[tempIdx] - *avg) / *deviation;
        }
        idx += blockDim.x * gridDim.x;
    }
}

int main(int argc, char* argv[])
{
    vector<pair<string, vector<double>> > out;
    out = readFromCsv("dataset/bigheartdata.csv");
    standardize("standarized_cuda.csv" , out);
}


void standardize(string outFile, vector< pair<string, vector<double>> >& data) {
    int i;
    int size = data[0].second.size();
    double * tempData = new double[size];
    for ( i = 0; i < data.size(); ++i) {
        if (data[i].first != "target") {
            double avg = findAverage(data[i].second);
            double deviation = findDeviation(data[i].second);
            //copy vector to array
            for (int j = 0; j < size; ++j){
                tempData[j] = data[i].second[j];
            }

            //creating cuda variables and memory allocation
            double * cudaData;
		    double * cudaAvg;
            double * cudaDeviation;
		    int * dataSize;
            hipMalloc( (void**)&cudaData, sizeof(double) * size ) ;    
    	    hipMalloc( (void**)&cudaAvg, sizeof(double));
    	    hipMalloc( (void**)&cudaDeviation, sizeof(double));
		    hipMalloc( (void**)&dataSize, sizeof(int));
           
            hipMemcpy( dataSize, &size, sizeof(int), hipMemcpyHostToDevice);
  	        hipMemcpy( cudaData, tempData, sizeof(double) * size, hipMemcpyHostToDevice );
    	    hipMemcpy( cudaDeviation, &deviation, sizeof(double), hipMemcpyHostToDevice );
    	    hipMemcpy( cudaAvg, &avg, sizeof(double), hipMemcpyHostToDevice );
		
            //standardization
            int num_blocks = ceil(size /1000) + 1;
            standarizeCuda<<<num_blocks, 1000>>>(cudaData, cudaDeviation, cudaAvg, dataSize);

            hipMemcpy( tempData, cudaData, sizeof(double) * size, hipMemcpyDeviceToHost );

            //copy table to vector 
            for (int j = 0; j < size; ++j){
                data[i].second[j] = tempData[j];
            }
           
        }
    }

    // zapisywanie do pliku
    // writeToCsv(outFile, data);
}

double findAverage(vector<double> data) {
    double average = accumulate(data.begin(), data.end(), 0.0) / data.size();
    return average;
}

// https://www.statisticshowto.com/probability-and-statistics/standard-deviation/#HFSSD - pattern
double findDeviation(vector<double> data) {
    double sum = accumulate(data.begin(), data.end(), 0.0);
	
    double square = sum * sum / data.size();
    double squareSum = 0;

    for (int i = 0; i < data.size(); ++i) {
        squareSum += data[i] * data[i];
    }

    double diff = squareSum - square;
    double variance = diff / (data.size() - 1);
    double deviation = sqrt(variance);

    return deviation;
}
