#include "hip/hip_runtime.h"
#include "CsvFunctions.h"
#include <algorithm> 

__global__ void distanceForEuclidean(double *data, double *test, int *size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    while (idx < *size) {
        if (idx < *size){
            int tempIdx = idx % 13;
			double diff = data[idx] - test[tempIdx];
            data[idx] = diff * diff;
        }
        idx += blockDim.x * gridDim.x;
    }
}

class Knn {
private:
	int k_numbers;
	int metric;
	int targetColumn;
	int threadNum;
public:
	vector<vector<double>> trainData;
	vector<vector<double>> learningData;
	Knn(int k = 1, int m = 1, int thrd = 1) {
		k_numbers = k;
		metric = m;
		targetColumn = 0;
		threadNum = thrd;
	}

	void setMetric(int number) {
		metric = number;
	}

	void setK(int k) {
		k_numbers = k;
	}

	void loadData(string file, int targetColumnNumber, int trainingPercent = 30) {
		targetColumn = targetColumnNumber - 1;
		vector<vector<double>> data = readFromCsvWithoutLabels(file);
		std::random_shuffle(data.begin(), data.end());
		int startIndex = (trainingPercent / 100.0) * data.size();
		vector<vector<double>> train(data.end() - startIndex, data.begin() + data.size());
		data.erase(data.end() - startIndex, data.begin() + data.size());
		learningData = data;
		trainData = train;
	}

	int predict(vector<double> features) {
		vector<pair<double, int>> distancesAndLabels = {};
		int size = learningData.size() * (learningData[0].size() - 1);
		double * tempData = new double[size];
		double * tempLabels = new double[learningData.size()];
		double * tempTestData = new double[learningData[0].size() - 1];
		for (int i = 0; i < learningData.size(); ++i){
			for( int j = 0; j < learningData[0].size(); ++j){
				if (j != 13) {
					tempData[i*13+j] = learningData[i][j];
				} else {
					tempLabels[i] = learningData[i][j];
				}
			}
		}

		for (int i = 0; i < features.size(); ++i) {
			tempTestData[i] = features[i];
		}
		
		double * cudaData;
		double * cudaTestValues;
		int * dataSize;
		hipMalloc( (void**)&cudaData, sizeof(double) * size ) ;
    	hipMalloc( (void**)&cudaTestValues, sizeof(double) * 13);
		hipMalloc( (void**)&dataSize, sizeof(int));
		hipMemcpy( dataSize, &size, sizeof(int), hipMemcpyHostToDevice);
  	    hipMemcpy( cudaData, tempData, sizeof(double) * size, hipMemcpyHostToDevice );
    	hipMemcpy( cudaTestValues, tempTestData, sizeof(double) * 13, hipMemcpyHostToDevice );
		int num_blocks = ceil(size /1000) + 1;
		distanceForEuclidean<<<num_blocks, 1000>>>(cudaData, cudaTestValues, dataSize);
    	hipMemcpy( tempData, cudaData, sizeof(double) * size, hipMemcpyDeviceToHost );
 		hipFree( cudaData );
    	hipFree( cudaTestValues );
		hipDeviceSynchronize();
		
		for (int i = 0; i < learningData.size(); ++i) {
		 	double sum = 0;
		  	for( int j = 0; j < learningData[0].size() - 1; ++j){
		 		 sum += tempData[i*13 + j];
		 	 }
		 	double euclidean = sqrt(sum);
		 	distancesAndLabels.push_back({ sum, tempLabels[i] });
	    }

	    sort(distancesAndLabels.begin(), distancesAndLabels.end());
	    vector<int> nearestResults = {0, 0};

		for (int i = 0; i < k_numbers; ++i) {
		  	nearestResults[(int)distancesAndLabels[i].second]++;
		}

		delete tempData;
		delete tempLabels;
		delete tempTestData;

		if (nearestResults[0] > nearestResults[1]) {
		    return 0;
		}
		else {
		    return 1;
		}

	}

	double checkAccuracy() {
		int good = 0;
		int bad = 0;
	
		for (int i = 0; i < 10; ++i) {
			int predictedTarget = predict(trainData[i]);
			if (predictedTarget == trainData[i][targetColumn]) {
				++good;
			}
			else {
				++bad;
			}
		}
		return good / (double)(good + bad);
	}

	double euclideanDistance(vector<double> learning, vector<double> target) {
		vector<double> distanceSquares = {};
		double euclideanDistance = 0;
		for (int i = 0; i < learning.size(); ++i) {
			if (i != targetColumn) {
				double diff = learning[i] - target[i];
				distanceSquares.push_back(diff * diff);
			}
		}
		
		for (int i = 0; i < distanceSquares.size(); ++i) {
			euclideanDistance += distanceSquares[i];
		}

		euclideanDistance = sqrt(euclideanDistance);
		return euclideanDistance;
	}
};


int main(int argc, char* argv[]) {
	Knn* knn = new Knn(5,0);
	knn->loadData("dataset/bigheartdata.csv", 14, 30);
	double accuracy = knn->checkAccuracy();
	cout << endl << accuracy;
	delete knn;
}
